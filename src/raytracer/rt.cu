#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <cudaGL.h>

#include "rt.cuh"

#include <cassert>
#include <cstdio>

static bool CUDA_check_error(const char * errMsg)
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s (%s)\n", errMsg, hipGetErrorString(err));
        return true;
    }
    return false;
}

bool CUDA_device_info()
{
    hipDeviceProp_t deviceProp = cudaDevicePropDontCare;
    int devCount = 0;
    hipGetDeviceCount(&devCount);
    if (CUDA_check_error("failed to get device count")) {
        return false;
    }
    fprintf(stderr, "CUDA: device count: %i\n", devCount);
    for (int dev = 0; dev < devCount; ++dev) {
        hipGetDeviceProperties(&deviceProp, dev);
        if (CUDA_check_error("failed to get device properties")) {
            return false;
        }
        fprintf(stderr,
                "CUDA device %i properties:\n"
                "name: %.256s\n"
                "totalGlobalMem: %zu bytes\n"
                "sharedMemPerBlock: %zu bytes\n"
                "regsPerBlock: %i\n"
                "warpSize: %i threads\n"
                "memPitch: %zu bytes\n"
                "maxThreadsPerBlock: %i\n"
                "maxThreadsDim: (%i,%i,%i)\n"
                "maxGridSize: (%i,%i,%i)\n"
                "totalConstMem: %zu bytes\n"
                "major: %i\n"
                "minor: %i\n"
                "clockRate: %i kHz\n"
                "textureAlignment: %zu\n"
                "deviceOverlap: %i\n"
                "multiProcessorCount: %i\n"
                "kernelExecTimeoutEnabled: %i\n"
                "integrated: %i\n"
                "canMapHostMemory: %i\n"
                "computeMode: %i\n"
                "concurrentKernels: %i\n"
                "ECCEnabled: %i\n"
                "pciBusID: %i\n"
                "pciDeviceID: %i\n"
                "pciDomainID: %i\n"
                "tccDriver: %i\n"
                "asyncEngineCount: %i\n"
                "unifiedAddressing: %i\n"
                "memoryClockRate: %i kHz\n"
                "memoryBusWidth: %i bits\n"
                "l2CacheSize: %i bytes\n"
                "maxThreadsPerMultiProcessor: %i\n"
                "streamPrioritiesSupported: %i\n"
                "globalL1CacheSupported: %i\n"
                "localL1CacheSupported: %i\n"
                "sharedMemPerMultiprocessor: %zu bytes\n"
                "regsPerMultiprocessor: %i\n"
                "managedMemory: %i\n"
                "isMultiGpuBoard: %i\n"
                "multiGpuBoardGroupID: %i\n"
                "hostNativeAtomicSupported: %i\n"
                "singleToDoublePrecisionPerfRatio: %i\n"
                "pageableMemoryAccess: %i\n"
                "concurrentManagedAccess: %i\n"
                "computePreemptionSupported: %i\n"
                "canUseHostPointerForRegisteredMem: %i\n"
                "cooperativeLaunch: %i\n"
                "cooperativeMultiDeviceLaunch: %i\n"
                "sharedMemPerBlockOptin: %zu bytes\n"
                "%c",
                dev,
                deviceProp.name + 0,
                deviceProp.totalGlobalMem,
                deviceProp.sharedMemPerBlock,
                deviceProp.regsPerBlock,
                deviceProp.warpSize,
                deviceProp.memPitch,
                deviceProp.maxThreadsPerBlock,
                deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2],
                deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2],
                deviceProp.totalConstMem,
                deviceProp.major,
                deviceProp.minor,
                deviceProp.clockRate,
                deviceProp.textureAlignment,
                deviceProp.deviceOverlap,
                deviceProp.multiProcessorCount,
                deviceProp.kernelExecTimeoutEnabled,
                deviceProp.integrated,
                deviceProp.canMapHostMemory,
                deviceProp.computeMode,
                deviceProp.concurrentKernels,
                deviceProp.ECCEnabled,
                deviceProp.pciBusID,
                deviceProp.pciDeviceID,
                deviceProp.pciDomainID,
                deviceProp.tccDriver,
                deviceProp.asyncEngineCount,
                deviceProp.unifiedAddressing,
                deviceProp.memoryClockRate,
                deviceProp.memoryBusWidth,
                deviceProp.l2CacheSize,
                deviceProp.maxThreadsPerMultiProcessor,
                deviceProp.streamPrioritiesSupported,
                deviceProp.globalL1CacheSupported,
                deviceProp.localL1CacheSupported,
                deviceProp.sharedMemPerMultiprocessor,
                deviceProp.regsPerMultiprocessor,
                deviceProp.managedMemory,
                deviceProp.isMultiGpuBoard,
                deviceProp.multiGpuBoardGroupID,
                deviceProp.hostNativeAtomicSupported,
                deviceProp.singleToDoublePrecisionPerfRatio,
                deviceProp.pageableMemoryAccess,
                deviceProp.concurrentManagedAccess,
                deviceProp.computePreemptionSupported,
                deviceProp.canUseHostPointerForRegisteredMem,
                deviceProp.cooperativeLaunch,
                deviceProp.cooperativeMultiDeviceLaunch,
                deviceProp.sharedMemPerBlockOptin,
                '\n');
    }
    return true;
}

bool CUDA_init()
{
    int dev = -1;
    hipDeviceProp_t deviceProp = cudaDevicePropDontCare;
    deviceProp.canMapHostMemory = 1;
    hipChooseDevice(&dev, &deviceProp);
    if (CUDA_check_error("failed to choose device")) {
        return false;
    }
    assert(!(dev < 0));
    fprintf(stderr, "CUDA: device %i is chosen\n", dev);
    hipGetDeviceProperties(&deviceProp, dev);
    if (CUDA_check_error("failed to get device properties")) {
        return false;
    }
    if (!deviceProp.canMapHostMemory) {
        fprintf(stderr, "CUDA: device is unable to map host memory\n");
        return false;
    }
    hipSetDeviceFlags(hipDeviceMapHost);
    if (CUDA_check_error("failed to set device flags")) {
        return false;
    }
    hipSetDevice(dev);
    if (CUDA_check_error("failed to set device")) {
        return false;
    }
    return true;
}

void * CUDA_registerGLBuffer(GLuint glBuf)
{
    hipGraphicsResource_t cudaBuf = nullptr;
    hipGraphicsGLRegisterBuffer(&cudaBuf, glBuf, hipGraphicsRegisterFlagsNone);
    if (CUDA_check_error("failed to register buffer")) {
        return nullptr;
    }
    return cudaBuf;
}

bool CUDA_unregisterGLBuffer(void * cudaBuf)
{
    hipGraphicsUnregisterResource((hipGraphicsResource_t)cudaBuf);
    if (CUDA_check_error("failed to unregister resource for buffer")) {
        return false;
    }
    return true;
}

__global__ void run(float3 * buf, uint * scene, int w, int h)
{
    int x = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    int y = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;
    int index = w * y + x;
    int resolution = w * h;
    if (!(index < resolution)) {
        return;
    }
    float3 & p = buf[index];
    if (scene) {
        if ((scene[0] != 1) || (scene[1] != 8) || (scene[2] != 0) || (scene[3] != 42)) {
            p = {1.0f, 0.0f, 0.0f};
        } else {
            p = {0.0f, 0.0f, 1.0f};
        }
            /*
        switch ((index * 3) / resolution) {
        case 0 : p = {1.0f, 0.0f, 0.0f}; break;
        case 1 : p = {0.0f, 1.0f, 0.0f}; break;
        case 2 : p = {0.0f, 0.0f, 1.0f}; break;
        default : p = {0.0f, 0.0f, 0.0f};
        }*/
    } else {
        p = {1.0f, 1.0f, 1.0f};
    }
}

inline
int divUp(int dividend, int divisor)
{
    return (dividend + (divisor - 1)) / divisor;
}

void * CUDA_registerBuffer(void * f, std::size_t size)
{
    hipHostRegister(f, size, hipHostRegisterDefault);
    if (CUDA_check_error("unable to register host memory")) {
        return {};
    }
    void * p = {};
    hipHostGetDevicePointer(&p, f, 0);
    if (CUDA_check_error("unable to get device pointer for registered host memory")) {
        CUDA_unregisterBuffer(f);
        return {};
    }
    return p;
}

bool CUDA_unregisterBuffer(void * f)
{
    hipHostUnregister(f);
    if (CUDA_check_error("unable to unregister registered host memory")) {
        return false;
    }
    return true;
}

bool CUDA_render(void * cudaBuf, void * scene, int w, int h)
{
    hipGraphicsMapResources(1, (hipGraphicsResource_t *)&cudaBuf);
    if (CUDA_check_error("failed to map resource")) {
        return false;
    }
    void * devPtr = nullptr;
    std::size_t size = 0;
    hipGraphicsResourceGetMappedPointer(&devPtr, &size, (hipGraphicsResource_t)cudaBuf);
    if (CUDA_check_error("failed to get device pointer")) {
        return false;
    }
    assert(w * h * 3 * sizeof(GLfloat) == size);
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(divUp(w, threadsPerBlock.x), divUp(h, threadsPerBlock.y));
    if (numBlocks.x * numBlocks.y * numBlocks.z > 0) {
        run<<< numBlocks, threadsPerBlock >>>(static_cast< float3 * >(devPtr), static_cast< uint * >(scene), w, h);
        hipDeviceSynchronize();
        CUDA_check_error("failed to launch run() kernel");
    }
    hipGraphicsUnmapResources(1, (hipGraphicsResource_t *)&cudaBuf);
    if (CUDA_check_error("failed to unmap resource")) {
        return false;
    }
    return true;
}
